#include "hip/hip_runtime.h"
// main.cu

// to compile: nvcc -o main main.cu -lsfml-graphics -lsfml-window -lsfml-system

#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <random>
#include <array>
#include <iostream>

// Screen settings
const int width = 2560;
const int height = 1440;
const int num_particles = 50000;
const int num_green = num_particles / 3;
const int num_red = num_particles / 3;
const int num_yellow = num_particles - num_green - num_red;

// Particle structure
struct Particle {
    float x, y, vx, vy;
    unsigned char r, g, b, a;
};

// Host-side struct for rendering
struct HostParticle {
    float x, y;
    sf::Color color;
};

// CUDA kernel to apply interaction and update in one go
__global__ void update_particles_combined(
    Particle* particles,
    int num_particles,
    const float* interaction_matrix,
    int group_offset,
    int group_size,
    int* group_ids
) {
    extern __shared__ Particle shared[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= group_size) return;

    Particle& p1 = particles[group_offset + idx];
    float fx = 0, fy = 0;

    for (int g = 0; g < 3; g++) {
        int target_offset = group_ids[g];
        int target_size = group_ids[g + 3];

        for (int j = 0; j < target_size; ++j) {
            Particle p2 = particles[target_offset + j];
            float dx = p1.x - p2.x;
            float dy = p1.y - p2.y;
            float dist2 = dx * dx + dy * dy;

            if (dist2 > 0 && dist2 < 6400) {
                float dist = sqrtf(dist2);
                float interaction = interaction_matrix[group_ids[6] * 3 + g];
                float force = interaction / dist;

                fx += force * dx;
                fy += force * dy;
            }
        }
    }

    // Velocity update and damping
    p1.vx = (p1.vx + fx) * 0.5f;
    p1.vy = (p1.vy + fy) * 0.5f;

    // Position update
    p1.x += p1.vx;
    p1.y += p1.vy;

    // Bounce and clamp
    if (p1.x <= 0 || p1.x >= width) p1.vx *= -1;
    if (p1.y <= 0 || p1.y >= height) p1.vy *= -1;
    p1.x = fminf(fmaxf(p1.x, 0), width);
    p1.y = fminf(fmaxf(p1.y, 0), height);
}

// Function to create particles
void create_particles(std::vector<Particle>& particles, int number, sf::Color color) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis_x(50, width - 50);
    std::uniform_real_distribution<float> dis_y(50, height - 50);
    std::uniform_real_distribution<float> dis_v(-1.0f, 1.0f);

    for (int i = 0; i < number; ++i) {
        Particle p;
        p.x = dis_x(gen);
        p.y = dis_y(gen);
        p.vx = dis_v(gen);
        p.vy = dis_v(gen);
        p.r = color.r;
        p.g = color.g;
        p.b = color.b;
        p.a = color.a;
        particles.push_back(p);
    }
}

int main() {
    sf::RenderWindow window(sf::VideoMode(width, height), "CUDA Particle Sim");

    std::vector<Particle> all_particles;
    create_particles(all_particles, num_green, sf::Color::Green);
    create_particles(all_particles, num_red, sf::Color::Red);
    create_particles(all_particles, num_yellow, sf::Color::Yellow);

    Particle* device_particles;
    hipMalloc(&device_particles, all_particles.size() * sizeof(Particle));
    hipMemcpy(device_particles, all_particles.data(), all_particles.size() * sizeof(Particle), hipMemcpyHostToDevice);

    float interaction_matrix[9];
    for (int i = 0; i < 9; ++i) interaction_matrix[i] = 0.0f;
    interaction_matrix[0] = 0.5f; // green-green
    interaction_matrix[4] = 0.5f; // red-red
    interaction_matrix[8] = 0.5f; // yellow-yellow


    float* device_matrix;
    hipMalloc(&device_matrix, sizeof(interaction_matrix));
    hipMemcpy(device_matrix, interaction_matrix, sizeof(interaction_matrix), hipMemcpyHostToDevice);

    int group_ids[7] = {
        0, num_green, num_green + num_red,        // group offsets
        num_green, num_red, num_yellow,           // group sizes
        0                                          // current group index (updated per loop)
    };
    int* device_group_ids;
    hipMalloc(&device_group_ids, sizeof(group_ids));

    int threads_per_block = 256;

    while (window.isOpen()) {
        sf::Event e;
        while (window.pollEvent(e)) {
            if (e.type == sf::Event::Closed)
                window.close();
        }

        for (int group = 0; group < 3; ++group) {
            group_ids[6] = group;  // current group index
            hipMemcpy(device_group_ids, group_ids, sizeof(group_ids), hipMemcpyHostToDevice);
            int offset = group_ids[group];
            int size = group_ids[group + 3];
            int blocks = (size + threads_per_block - 1) / threads_per_block;
            update_particles_combined<<<blocks, threads_per_block>>>(
                device_particles, all_particles.size(), device_matrix, offset, size, device_group_ids
            );
        }

        hipDeviceSynchronize();  // Wait for GPU work to finish

        hipMemcpy(all_particles.data(), device_particles, all_particles.size() * sizeof(Particle), hipMemcpyDeviceToHost);

        window.clear();

        sf::VertexArray vertices(sf::Points, all_particles.size());
        for (size_t i = 0; i < all_particles.size(); ++i) {
            vertices[i].position = sf::Vector2f(all_particles[i].x, all_particles[i].y);
            vertices[i].color = sf::Color(all_particles[i].r, all_particles[i].g, all_particles[i].b, all_particles[i].a);
        }

        window.draw(vertices);
        window.display();
    }

    hipFree(device_particles);
    hipFree(device_matrix);
    hipFree(device_group_ids);
    return 0;
}
